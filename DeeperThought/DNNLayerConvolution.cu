#include "hip/hip_runtime.h"
#include "DNNLayerConvolution.cuh"

#include <cstdlib>
#include <cstdio>

#define MAXX1X2 1000
#define MAXNUMCONVY1Y2 1000

__global__ void convolution_forward(float *outp, const float *inp, const float *pars, int numPics, int inputWidth, int outputWidth, int numConvolutions, int x1, int x2, int y1, int y2, int batchSize)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	if (tid < batchSize)
	{
		int c = tid % numConvolutions;
		tid = tid / numConvolutions;
		int p = tid % numPics;
		tid = tid / numPics;

		float pics[MAXX1X2];
		for (int i = 0; i < x1 * x2; i++)
		{
			pics[i] = inp[tid * inputWidth + p * x1 * x2 + i];
		}
		float convos[MAXNUMCONVY1Y2];
		for (int i = 0; i < y1 * y2; i++)
		{
			convos[i] = pars[c * y1 * y2 + i];
		}

		int pos = p * numConvolutions * (x1 - y1 + 1) * (x2 - y2 + 1) +  c * (x1 - y1 + 1) * (x2 - y2 + 1);
		for (int i = 0; i < x1 - y1 + 1; i++)
		{
			for (int j = 0; j < x2 - y2 + 1; j++)
			{
				float tmp = 0;
				for (int k = 0; k < y1; k++)
				{
					for (int l = 0; l < y2; l++)
					{
						tmp += pics[(i + k) * x2 + (j + l)] * convos[k * y2 + l];
					}
				}
				outp[tid * outputWidth + pos] = tmp;
				pos++;
			}
		}
	}
}

__global__ void convolution_backward(float *dinp, float *dpars, const float *doutp, const float *outp, const float *inp, const float *pars, int numPics, int inputWidth, int outputWidth, int numConvolutions, int x1, int x2, int y1, int y2, int batchSize)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	if (tid < batchSize)
	{
		int c = tid % numConvolutions;
		tid = tid / numConvolutions;
		int p = tid % numPics;
		tid = tid / numPics;

		float pics[MAXX1X2];
		for (int i = 0; i < x1 * x2; i++)
		{
			pics[i] = inp[tid * inputWidth + p * x1 * x2 + i];
		}
		float convos[MAXNUMCONVY1Y2];
		for (int i = 0; i < y1 * y2; i++)
		{
			convos[i] = pars[c * y1 * y2 + i];
		}

		int pos = p * numConvolutions * (x1 - y1 + 1) * (x2 - y2 + 1) + c * (x1 - y1 + 1) * (x2 - y2 + 1);
		for (int i = 0; i < x1 - y1 + 1; i++)
		{
			for (int j = 0; j < x2 - y2 + 1; j++)
			{
				float tmp = doutp[tid * outputWidth + pos];
				if (tmp != 0)
				{
					for (int k = 0; k < y1; k++)
					{
						for (int l = 0; l < y2; l++)
						{
							if (dinp != NULL)
							{
								atomicAdd(&(dinp[tid * inputWidth + p * x1 * x2 + (i + k) * x2 + (j + l)]), tmp * convos[k * y2 + l]);
							}
							atomicAdd(&(dpars[c * y1 * y2 + k * y2 + l]), tmp * pics[(i + k) * x2 + (j + l)]);
						}
					}
				}
				pos++;
			}
		}
	}
}

DNNLayerConvolution::DNNLayerConvolution(int _numPics, int _x1, int _x2, int _numConvolutions, int _y1, int _y2, int _batchSize, float _initVal, float _stepSize)
	: DNNLayer(_batchSize, _numPics * _x1 * _x2, _numPics * (_x1 - _y1 + 1) * (_x2 - _y2 + 1) * _numConvolutions, _numConvolutions * _y1 * _y2, _initVal, _stepSize)
{
	x1 = _x1;
	x2 = _x2;
	y1 = _y1;
	y2 = _y2;
	numPics = _numPics;
	numConvolutions = _numConvolutions;

	if (x1 * x2 > MAXX1X2)
	{
		fprintf(stderr, "Project needs to be recompiled with larger field for convolution layer\n");
		exit(-1);
	}
	if (y1 * y2 > MAXNUMCONVY1Y2)
	{
		fprintf(stderr, "Project needs to be recompiled with larger field for convolution layer\n");
		exit(-1);
	}
}

DNNLayerConvolution::~DNNLayerConvolution()
{

}

void DNNLayerConvolution::Forward(CPUGPUMemory* input)
{
	int threadsPerBlock = 256;
	int numBlocks = ((input->GetSize() / inputWidth) * numConvolutions * numPics + threadsPerBlock - 1) / threadsPerBlock;
	convolution_forward<<<numBlocks, threadsPerBlock>>>(
		(float*)output->GetGPUMemory(), (float*)input->GetGPUMemory(), (float*)params->GetGPUMemory(), numPics, inputWidth, outputWidth, numConvolutions, x1, x2, y1, y2, (input->GetSize() / inputWidth) * numConvolutions * numPics);
}

void DNNLayerConvolution::Backward(CPUGPUMemory* input, CPUGPUMemory* deltaOutput)
{
	int threadsPerBlock = 256;
	int numBlocks = ((input->GetSize() / inputWidth) * numConvolutions * numPics + threadsPerBlock - 1) / threadsPerBlock;
	convolution_backward<<<numBlocks, threadsPerBlock>>>(deltaInput == NULL ? NULL : (float*)deltaInput->GetGPUMemory(), (float*)dparams->GetGPUMemory(), (float*)deltaOutput->GetGPUMemory(),
		(float*)output->GetGPUMemory(), (float*)input->GetGPUMemory(), (float*)params->GetGPUMemory(), numPics, inputWidth, outputWidth, numConvolutions, x1, x2, y1, y2, (input->GetSize() / inputWidth) * numConvolutions * numPics);
}
