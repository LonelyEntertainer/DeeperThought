#include "hip/hip_runtime.h"
#include "DNNLayerMovieUser.cuh"

#include <cstdlib>
#include <cstdio>
#include <cfloat>

#define MAXVECWIDTH 1000

__global__ void movieuser_forward(float *outp, const int *inp, const float *params, int numUsers, int numMovies, int vectorWidthMovie, int vectorWidthUser, int batchSize)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	if (tid < batchSize)
	{
		float movie[MAXVECWIDTH];
		float user[MAXVECWIDTH];

		int m = inp[2 * tid];
		int u = inp[2 * tid + 1];

		for (int i = 0; i < vectorWidthMovie; i++)
		{
			movie[i] = params[m * vectorWidthMovie + i];
		}
		for (int i = 0; i < vectorWidthUser; i++)
		{
			user[i] = params[numMovies *vectorWidthMovie + u * vectorWidthUser + i];
		}

		for (int i = 0; i < vectorWidthMovie; i++)
		{
			for (int j = 0; j < vectorWidthUser; j++)
			{
				outp[tid * vectorWidthMovie * vectorWidthUser + i * vectorWidthUser + j] = movie[i] * user[j];
			}
		}
	}
}

__global__ void movieuser_backward(float *dparams, const float *doutp, const float *outp, const int *inp, const float *params, int numUsers, int numMovies, int vectorWidthMovie, int vectorWidthUser, int batchSize)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	if (tid < batchSize)
	{
		float movie[MAXVECWIDTH];
		float user[MAXVECWIDTH];
		float dmovie[MAXVECWIDTH];
		float duser[MAXVECWIDTH];

		int m = inp[2 * tid];
		int u = inp[2 * tid + 1];

		for (int i = 0; i < vectorWidthMovie; i++)
		{
			movie[i] = params[m * vectorWidthMovie + i];
			dmovie[i] = 0;
		}
		for (int i = 0; i < vectorWidthUser; i++)
		{
			user[i] = params[numMovies *vectorWidthMovie + u * vectorWidthUser + i];
			duser[i] = 0;
		}

		for (int i = 0; i < vectorWidthMovie; i++)
		{
			for (int j = 0; j < vectorWidthUser; j++)
			{
				dmovie[i] += doutp[tid * vectorWidthMovie * vectorWidthUser + i * vectorWidthUser + j] * user[j];
				duser[j] += doutp[tid * vectorWidthMovie * vectorWidthUser + i * vectorWidthUser + j] * movie[i];
			}
		}

		for (int i = 0; i < vectorWidthMovie; i++)
		{
			atomicAdd(&(dparams[m * vectorWidthMovie + i]), dmovie[i]);
		}
		for (int i = 0; i < vectorWidthUser; i++)
		{
			atomicAdd(&(dparams[numMovies * vectorWidthMovie + u * vectorWidthUser + i]), duser[i]);
		}
	}
}

DNNLayerMovieUser::DNNLayerMovieUser(int _numMovies, int _numUsers, int _vectorWidthMovie, int _vectorWidthUser, float _initValues, float _stepSize, int _batchSize)
	: DNNLayer(_batchSize, 2, _vectorWidthMovie * _vectorWidthUser, _numMovies * _vectorWidthMovie + _numUsers * _vectorWidthUser, _initValues, _stepSize)
{
	numMovies = _numMovies;
	numUsers = _numUsers;
	vectorWidthMovie = _vectorWidthMovie;
	vectorWidthUser = _vectorWidthUser;
}

DNNLayerMovieUser::~DNNLayerMovieUser()
{

}

void DNNLayerMovieUser::Forward(CPUGPUMemory* input)
{
	int threadsPerBlock = 256;
	int numBlocks = ((input->GetSize() / inputWidth) + threadsPerBlock - 1) / threadsPerBlock;
	movieuser_forward << <numBlocks, threadsPerBlock >> >(
		(float*)output->GetGPUMemory(), (int*)input->GetGPUMemory(), (float*)params->GetGPUMemory(), numUsers, numMovies, vectorWidthMovie, vectorWidthUser, (input->GetSize() / inputWidth));
}

void DNNLayerMovieUser::Backward(CPUGPUMemory* input, CPUGPUMemory* deltaOutput)
{
	int threadsPerBlock = 256;
	int numBlocks = ((input->GetSize() / inputWidth) + threadsPerBlock - 1) / threadsPerBlock;
	movieuser_backward << <numBlocks, threadsPerBlock >> > ((float*)dparams->GetGPUMemory(), (float*)deltaOutput->GetGPUMemory(),
		(float*)output->GetGPUMemory(), (int*)input->GetGPUMemory(), (float*)params->GetGPUMemory(), numUsers, numMovies, vectorWidthMovie, vectorWidthUser, (input->GetSize() / inputWidth));
}
